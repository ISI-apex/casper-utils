
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
	for(int i = 0; i < n; i++){
		out[i] = a[i] + b[i];
	}
}

int main(){
	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	// Allocate memory
	a   = (float*)malloc(sizeof(float) * N);
	b   = (float*)malloc(sizeof(float) * N);
	out = (float*)malloc(sizeof(float) * N);

	hipMalloc((void **)&d_a, sizeof(float) * N);
	hipMalloc((void **)&d_b, sizeof(float) * N);
	hipMalloc((void **)&d_out, sizeof(float) * N);
	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	// Initialize array
	for(int i = 0; i < N; i++){
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	// Main function
	vector_add<<<1,1>>>(d_out, d_a, d_b, N);
	hipDeviceSynchronize();

	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);

	// Verification
	for(int i = 0; i < N; i++){
		printf("%f %f %f\n", a[i], b[i], out[i]);
		assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
	}

	printf("out[0] = %f\n", out[0]);
	printf("PASSED\n");
}
